
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <unistd.h>

// **********************************************
// For floats vector on device
// **********************************************

typedef struct {
  float x;
  float y;
  float z;
} Vec;

__device__ Vec vec_add(Vec v1, Vec v2) {
  Vec res;
  res.x = v1.x + v2.x;
  res.y = v1.y + v2.y;
  res.z = v1.z + v2.z;
  return res;
}

__device__ Vec vec_sub(Vec v1, Vec v2) {
  Vec res;
  res.x = v1.x - v2.x;
  res.y = v1.y - v2.y;
  res.z = v1.z - v2.z;
  return res;
}

__device__ Vec vec_scale(Vec v, float i) {
  Vec res;
  res.x = i * v.x;
  res.y = i * v.y;
  res.z = i * v.z;
  return res;
}

__device__ float random_float(uint64_t * seed){
  const uint64_t m = 9223372036854775808ULL;
  const uint64_t a = 2806196910506780709ULL;
  const uint64_t c = 1ULL;
  *seed = (a * (*seed) + c)%m;
  float res = (float) (*seed)/(float)m;
  return res;
}
  
__device__ uint64_t forward(uint64_t seed, uint64_t n){
  const uint64_t m = 9223372036854775808ULL;
  uint64_t a = 2806196910506780709ULL;
  uint64_t c = 1ULL;
  n = n % m;
  uint64_t a_new = 1;
  uint64_t c_new = 0;
  while(n>0){
    if(n & 1){
      a_new *= a;
      c_new = c_new *a + c;
    }
    c *= (a + 1);
    a *= a;
    n >>= 1;
  }
  return (a_new * seed + c_new) % m;
}

__device__ Vec vec_sample_unit(uint64_t i) {
  Vec res;
  uint64_t seed = forward(i, i*200);
  float pho = random_float(&seed) * 2 * 3.141592653;   
  float cos_theta = random_float(&seed) * 2 - 1;
  float sin_theta = sqrt( 1 - cos_theta * cos_theta);
  res.x = sin_theta * cos(pho);
  res.y = sin_theta * sin(pho);
  res.z = cos_theta;
  return res;
}

__device__ float vec_dot_product(Vec v1, Vec v2) {
  float res;
  res = v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
  return res;
}

__device__ float vec_norm(Vec v) {
  float res;
  res = sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
  return res;
}

// **********************************************
// ray_trace_kernel
// **********************************************
inline void cuchk(hipError_t err){
  if (err != hipSuccess) {
     printf("%s in %s at line %d\n",
     	hipGetErrorString(err), __FILE__, __LINE__);
     exit(EXIT_FAILURE);
  }
}

__global__ void ray_trace_kernel(float* d_window, int width, int ray_num) {

  //tried to use the shared mem, but it's too small to store the whole window 1000*1000*4bytes = 4 mb  
  
  //issue the threads to do the work
  __shared__ int ray_chunck;
  __shared__ Vec C;
  __shared__ Vec L;
  __shared__ float w_y;
  __shared__ float w_max;
  __shared__ float R;

  int tx = blockIdx.x * gridDim.x + threadIdx.x;
  ray_chunck = ray_num/gridDim.x;
  C.x = 0.0;
  C.y = 12.0;
  C.z = 0.0;
  L.x = 4.0;
  L.y = 4.0;
  L.z = -1;
  w_y = 10;
  w_max = 10;
  R = 6.0;
  
  for (int i = 0; i < ray_chunck; i+=blockDim.x*gridDim.x){
    Vec V, W;
    uint64_t seed = tx;
    while(1){
      seed += ray_num;
      V = vec_sample_unit(seed);
      if (V.y == 0) continue;
      W = vec_scale(V, (w_y/V.y));
      float temp = vec_dot_product(V, C);
      if (fabs(W.x) < w_max && fabs(W.z) < w_max && temp * temp + R * R - vec_dot_product(C,C) > 0) break;
    }
    float temp2 = vec_dot_product(V,C);
    float t = temp2 - sqrt(temp2 * temp2 + R * R - vec_dot_product(C,C));
    Vec II = vec_scale(V, t);
    Vec N = vec_scale(vec_sub(II,C), 1.0/vec_norm(vec_sub(II,C)));
    Vec S = vec_scale(vec_sub(L,II), 1.0/vec_norm(vec_sub(L,II)));
    float b = vec_dot_product(S,N); 
    b = 0 >= b ? 0 : b;
    int x = floor((W.x + w_max) /(2 * w_max) * (width - 1));
    int z = floor((W.z + w_max) /(2 * w_max) * (width - 1));
    atomicAdd(&d_window[x*width + z], b);
  }
}

// **********************************************
// get the args
// **********************************************

void get_input(int argc, char *argv[], int* num_rays, int* len, int *grid_dim, int *block_dim){
  *num_rays = 1000000;
  *len = 1000;
  *grid_dim = -1;
  *block_dim = 256;
  int opt;
  while((opt = getopt(argc, argv, "r:l:g:b:")) != -1) {
    switch (opt) {
    case 'r':
      *num_rays = atoi(optarg);
      break;
    case 'l':
      *len = atoi(optarg);
      break;
    case 'g':
      *grid_dim = atoi(optarg);
      break;
    case 'b':
      *block_dim = atoi(optarg);
      break;
    default:break;
    }
  }
  if (*grid_dim == -1) {
    *grid_dim = (*num_rays + *block_dim-1) / *block_dim;
  }
}


int main(int argc, char* argv[]) {
  int num_rays;
  int len;
  int grid_dim;
  int block_dim;
  get_input(argc, argv, &num_rays, &len, &grid_dim, &block_dim);
  
  size_t size = len * len * sizeof(float);

  float* d_window;
  hipMalloc((void **) &d_window, size);
  hipMemset((void *) d_window, 0.0, size);
  
  hipEvent_t start ,end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  
  hipEventRecord(start, 0);
  ray_trace_kernel<<<grid_dim,block_dim>>>(d_window, len, num_rays);
  hipEventRecord(end, 0);

  float time;
  hipEventSynchronize(end);
  hipEventElapsedTime(&time, start, end);
  time = time / 1000;
  printf("ray_num\tgrid_dim\tblock_dim\ttime\n");
  printf("%d\t%d\t%d\t%f\n", num_rays, grid_dim, block_dim, time);

  float* window = (float*) malloc(size);
  hipMemcpy(window, d_window, size, hipMemcpyDeviceToHost);

  char filename[] = "ball.dat";
  FILE *f = fopen(filename, "wb");
  if (f != NULL) {
    fwrite(window, sizeof(float), len * len, f);
    fclose(f);
  } else {
    fprintf(stderr, "Error opening %s: ", filename);
    perror("");
    free(window);
    cuchk(hipFree(d_window));
    exit(EXIT_FAILURE);
  }

  free(window);
  cuchk(hipFree(d_window));
  return 0;
}
